#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <gmac.h>

#include "utils.h"
#include "debug.h"


const size_t vecSize = 1024 * 1024;
const size_t blockSize = 512;

const char *msg = "Done!";

__global__ void vecAdd(float *c, float *a, float *b, size_t size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i >= size) return;

    c[i] = a[i] + b[i];
}


float doTest(float *a, float *b, float *c, float *orig)
{
	struct timeval s, t;

    FILE * fA = fopen("inputset/vectorA", "r");
    FILE * fB = fopen("inputset/vectorB", "r");
    gettimeofday(&s, NULL);
    fread(a, sizeof(float), vecSize, fA);
    fread(b, sizeof(float), vecSize, fB);

    gettimeofday(&t, NULL);
    fclose(fA);
    fclose(fB);
    printTime(&s, &t, "Init: ", "\n");

    // Call the kernel
    gettimeofday(&s, NULL);
    dim3 Db(blockSize);
    dim3 Dg(vecSize / blockSize);
    if(vecSize % blockSize) Dg.x++;
    vecAdd<<<Dg, Db>>>(gmacPtr(c), gmacPtr(a), gmacPtr(b), vecSize);
    if(gmacThreadSynchronize() != gmacSuccess) CUFATAL();
    gettimeofday(&t, NULL);
    printTime(&s, &t, "Run: ", "\n");

    gettimeofday(&s, NULL);
    float error = 0;
    for(int i = 0; i < vecSize; i++) {
        error += orig[i] - (c[i]);
    }
    gettimeofday(&t, NULL);
    fprintf(stderr,"Error: %f\n", error);
    printTime(&s, &t, "Check: ", "\n");

    return error;
}

int main(int argc, char *argv[])
{
	float *a, *b, *c;
	struct timeval s, t;
    float error1, error2, error3;

	fprintf(stdout, "Vector: %f\n", 1.0 * vecSize / 1024 / 1024);

    float * orig = (float *) malloc(vecSize * sizeof(float));
    FILE * fO = fopen("inputset/vectorC", "r");
    fread(orig, sizeof(float), vecSize, fO);

    // Alloc output data
    if(gmacMalloc((void **)&c, vecSize * sizeof(float)) != gmacSuccess)
        CUFATAL();

    //////////////////////
    // Test shared objects
    //////////////////////
    fprintf(stderr,"SHARED OBJECTS\n");
    gettimeofday(&s, NULL);
    // Alloc & init input data
    if(gmacMalloc((void **)&a, vecSize * sizeof(float)) != gmacSuccess)
        CUFATAL();
    if(gmacMalloc((void **)&b, vecSize * sizeof(float)) != gmacSuccess)
        CUFATAL();
    gettimeofday(&t, NULL);
    printTime(&s, &t, "Alloc: ", "\n");

    error1 = doTest(a, b, c, orig);

    FILE * fC = fopen("vectorC_shared", "w");
    fwrite(c, sizeof(float), vecSize, fC);
    fclose(fC);

    gmacFree(a);
    gmacFree(b);

    //////////////////////////
    // Test replicated objects
    //////////////////////////
    fprintf(stderr,"REPLICATED OBJECTS\n");
    gettimeofday(&s, NULL);
    // Alloc & init input data
    if(gmacGlobalMalloc((void **)&a, vecSize * sizeof(float), GMAC_GLOBAL_MALLOC_REPLICATED) != gmacSuccess)
        CUFATAL();
    if(gmacGlobalMalloc((void **)&b, vecSize * sizeof(float), GMAC_GLOBAL_MALLOC_REPLICATED) != gmacSuccess)
        CUFATAL();
    gettimeofday(&t, NULL);
    printTime(&s, &t, "Alloc: ", "\n");

    error2 = doTest(a, b, c, orig);

    fC = fopen("vectorC_replicated", "w");
    fwrite(c, sizeof(float), vecSize, fC);
    fclose(fC);

    gmacFree(a);
    gmacFree(b);

    ///////////////////////////
    // Test centralized objects
    ///////////////////////////
    fprintf(stderr,"CENTRALIZED OBJECTS\n");
    gettimeofday(&s, NULL);
    // Alloc & init input data
    if(gmacGlobalMalloc((void **)&a, vecSize * sizeof(float), GMAC_GLOBAL_MALLOC_CENTRALIZED) != gmacSuccess)
        CUFATAL();
    if(gmacGlobalMalloc((void **)&b, vecSize * sizeof(float), GMAC_GLOBAL_MALLOC_CENTRALIZED) != gmacSuccess)
        CUFATAL();
    gettimeofday(&t, NULL);
    printTime(&s, &t, "Alloc: ", "\n");

    error3 = doTest(a, b, c, orig);

    fC = fopen("vectorC_centralized", "w");
    fwrite(c, sizeof(float), vecSize, fC);
    fclose(fC);

    gmacFree(a);
    gmacFree(b);

    gmacFree(c);
    free(orig);
    return error1 != 0.f && error2 != 0.f && error3 != 0.f;
}
