#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gmac.h>

const size_t size = 4 * 1024 * 1024;
const size_t blockSize = 512;

__global__ void reset(long *a, long v)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= size) return;
	a[i] += v;
}

int check(long *ptr, int s)
{
	int a = 0;
	for(int i = 0; i < size; i++)
		a += ptr[i];
	return a - s;
}

int main(int argc, char *argv[])
{
	long *ptr;
	assert(gmacMalloc((void **)&ptr, size * sizeof(long)) == gmacSuccess);

	// Call the kernel
	dim3 Db(blockSize);
	dim3 Dg(size / blockSize);
	if(size % blockSize) Db.x++;

	fprintf(stderr,"Test full memset: ");
	gmacMemset(ptr, 0, size * sizeof(long));
	reset<<<Dg, Db>>>(gmacPtr(ptr), 1);
	fprintf(stderr,"%d\n", check(ptr, size));

	fprintf(stderr, "Test partial memset: ");
	gmacMemset(&ptr[size / 8], 0, 3 * size / 4 * sizeof(long));
	fprintf(stderr,"%d\n", check(ptr, size / 4));

	gmacFree(ptr);

    return 0;
}
