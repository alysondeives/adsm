#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gmac.h>

const size_t size = 4 * 1024 * 1024;
const size_t blockSize = 512;

__global__ void reset(long *a, long v)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= size) return;
	a[i] += v;
}

int check(long *ptr, int s)
{
	int a = 0;
	for(int i = 0; i < size; i++)
		a += ptr[i];
	return a - s;
}

int main(int argc, char *argv[])
{
	long *ptr;
	assert(gmacMalloc((void **)&ptr, size * sizeof(long)) == gmacSuccess);

	// Call the kernel
	dim3 Db(blockSize);
	dim3 Dg(size / blockSize);
	if(size % blockSize) Db.x++;

	fprintf(stderr,"Test full memset: ");
	//gmacMemset(ptr, 0, size * sizeof(long));
	memset(ptr, 1, size * sizeof(long));
	memset(ptr, 0, size * sizeof(long));

    for (int i = 0; i < size; i++) {
        if (ptr[i] != 0) {
            printf("WTF %d\n", i);
        }
    }

	reset<<<Dg, Db>>>(gmacPtr(ptr), 1);
    gmacThreadSynchronize();
	fprintf(stderr,"%d\n", check(ptr, size));

    for (int i = 0; i < size; i++) {
        if (ptr[i] != 1) {
            printf("WTF2 %d\n", i);
        }
    }

	fprintf(stderr, "Test partial memset: ");
	//gmacMemset(&ptr[size / 8], 0, 3 * size / 4 * sizeof(long));
	memset(ptr, 0, size * sizeof(long));
    gmacThreadSynchronize();
	fprintf(stderr,"%d\n", check(ptr, size / 4));

	gmacFree(ptr);

    return 0;
}
