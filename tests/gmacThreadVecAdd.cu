#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#include <pthread.h>

#include <gmac.h>

#include "utils.h"
#include "debug.h"

const char *nIterStr = "GMAC_NITER";
const char *vecSizeStr = "GMAC_VECSIZE";

const unsigned nIterDefault = 2;
const size_t vecSizeDefault = 1024 * 1024;

unsigned nIter = 0;
size_t vecSize = 0;
const size_t blockSize = 512;

static float **s;

__global__ void vecAdd(float *c, float *a, float *b, size_t vecSize)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= vecSize) return;

	c[i] = a[i] + b[i];
}


void randInit(float *a, size_t vecSize)
{
	for(int i = 0; i < vecSize; i++) {
		a[i] = 1.0;
	}
}

void *addVector(void *ptr)
{
	float *a, *b;
	float **c = (float **)ptr;
	struct timeval s, t;
	gmacError_t ret = gmacSuccess;

	gettimeofday(&s, NULL);
	// Alloc & init input data
	ret = gmacMalloc((void **)&a, vecSize * sizeof(float));
	assert(ret == gmacSuccess);
	randInit(a, vecSize);
	ret = gmacMalloc((void **)&b, vecSize * sizeof(float));
	assert(ret == gmacSuccess);
	randInit(b, vecSize);

	// Alloc output data
	ret = gmacMalloc((void **)c, vecSize * sizeof(float));
	assert(ret == gmacSuccess);
	gettimeofday(&t, NULL);
	printTime(&s, &t, "Alloc: ", "\n");

	// Call the kernel
	dim3 Db(blockSize);
	dim3 Dg(vecSize / blockSize);
	if(vecSize % blockSize) Dg.x++;
	gettimeofday(&s, NULL);
	vecAdd<<<Dg, Db>>>(gmacPtr(*c), gmacPtr(a), gmacPtr(b), vecSize);
	if(gmacThreadSynchronize() != gmacSuccess) CUFATAL();
	gettimeofday(&t, NULL);
	printTime(&s, &t, "Run: ", "\n");

	gettimeofday(&s, NULL);
	float error = 0;
	for(int i = 0; i < vecSize; i++) {
		error += (*c)[i] - (a[i] + b[i]);
		//error += (a[i] - b[i]);
	}
	gettimeofday(&t, NULL);
	printTime(&s, &t, "Check: ", "\n");
	fprintf(stdout, "Error: %.02f\n", error);

	gmacFree(a);
	gmacFree(b);

	return NULL;
}


int main(int argc, char *argv[])
{
	pthread_t *nThread;
	unsigned n = 0;

	setParam<unsigned>(&nIter, nIterStr, nIterDefault);
	setParam<size_t>(&vecSize, vecSizeStr, vecSizeDefault);

	vecSize = vecSize / nIter;

	nThread = (pthread_t *)malloc(nIter * sizeof(pthread_t));
	s = (float **)malloc(nIter * sizeof(float **));

	srand(time(NULL));

	for(n = 0; n < nIter; n++) {
		pthread_create(&nThread[n], NULL, addVector, &s[n]);
	}

	for(n = 0; n < nIter; n++) {
		pthread_join(nThread[n], NULL);
	}

	float error = 0;
	for(n = 0; n < nIter; n++) {
		for(int i = 0; i < vecSize; i++) {
			error += s[n][i] - 2;
		}
	}
	fprintf(stdout, "Total: %.02f\n", error);

	free(s);
	free(nThread);

}
