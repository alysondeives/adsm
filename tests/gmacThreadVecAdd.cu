#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <pthread.h>

#include <gmac.h>
#include <gmac/paraver.h>

#include "debug.h"

const size_t vecSize = 1024 * 1024;
const size_t blockSize = 512;
const unsigned nIter = 2;

__global__ void vecAdd(float *c, float *a, float *b)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= vecSize) return;

	c[i] = a[i] + b[i];
}


void randInit(float *a, size_t vecSize)
{
	for(int i = 0; i < vecSize; i++) {
//		a[i] = rand() / (float)RAND_MAX;
		a[i] = 1.0;
	}
}

void *addVector(void *ptr)
{
	float *a, *b, *c;
	gmacError_t ret = gmacSuccess;

	// Alloc & init input data
	ret = gmacSafeMalloc((void **)&a, vecSize * sizeof(float));
	assert(ret == gmacSuccess);
	randInit(a, vecSize);
	ret = gmacSafeMalloc((void **)&b, vecSize * sizeof(float));
	assert(ret == gmacSuccess);
	randInit(b, vecSize);

	// Alloc output data
	ret = gmacSafeMalloc((void **)&c, vecSize * sizeof(float));
	assert(ret == gmacSuccess);

	// Call the kernel
	dim3 Db(blockSize);
	dim3 Dg(vecSize / blockSize);
	if(vecSize % blockSize) Db.x++;
	vecAdd<<<Dg, Db>>>(gmacSafe(c), gmacSafe(a), gmacSafe(b));
	if(gmacThreadSynchronize() != gmacSuccess) CUFATAL();

	float error = 0;
	for(int i = 0; i < vecSize; i++) {
		error += c[i] - (a[i] + b[i]);
		//error += (a[i] - b[i]);
	}
	fprintf(stdout, "Error: %.02f\n", error);

	gmacFree(a);
	gmacFree(b);
	gmacFree(c);

	return NULL;
}


int main(int argc, char *argv[])
{
	pthread_t nThread[nIter];
	unsigned n = 0;

	srand(time(NULL));

	for(n = 0; n < nIter; n++) {
		pthread_create(&nThread[n], NULL, addVector, NULL);
	}

	for(n = 0; n < nIter; n++) {
		pthread_join(nThread[n], NULL);
	}

}
