#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#include <pthread.h>

#include <gmac.h>

#include "utils.h"
#include "debug.h"

const unsigned rounds = 64;

const char *nIterStr = "GMAC_NITER";
const char *vecSizeStr = "GMAC_VECSIZE";

const unsigned nIterDefault = 1;
const size_t vecSizeDefault = 64 * 1024 * 1024;

unsigned nIter = 0;
size_t vecSize = 0;
const size_t blockSize = 512;

static float **s;

__global__ void vecAdd(float *c, float *a, float *b, size_t vecSize)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= vecSize) return;

	c[i] = a[i] + b[i];
}


void randInit(float *a, size_t vecSize)
{
	for(int i = 0; i < vecSize; i++) {
		a[i] = 1.0;
	}
}

void *addVector(void *ptr)
{
	float *a, *b;
	float **c = (float **)ptr;
	struct timeval s, t;
	gmacError_t ret = gmacSuccess;

	// Alloc & init input data
	ret = gmacMalloc((void **)&a, vecSize * sizeof(float));
	assert(ret == gmacSuccess);
	ret = gmacMalloc((void **)&b, vecSize * sizeof(float));
	assert(ret == gmacSuccess);

	gettimeofday(&s, NULL);
	for(int i = 0; i < rounds; i++) {
		randInit(a, vecSize);
		randInit(b, vecSize);
	}
	gettimeofday(&t, NULL);
	printAvgTime(&s, &t, "Alloc: ", "\n", rounds);

#if 0
	// Alloc output data
	ret = gmacMalloc((void **)c, vecSize * sizeof(float));
	assert(ret == gmacSuccess);

	// Call the kernel
	dim3 Db(blockSize);
	dim3 Dg(vecSize / blockSize);
	if(vecSize % blockSize) Dg.x++;
	gettimeofday(&s, NULL);
	vecAdd<<<Dg, Db>>>(gmacPtr(*c), gmacPtr(a), gmacPtr(b), vecSize);
	if(gmacThreadSynchronize() != gmacSuccess) CUFATAL();
	gettimeofday(&t, NULL);
	printTime(&s, &t, "Run: ", "\n");

	gettimeofday(&s, NULL);
	float error = 0;
	for(int i = 0; i < vecSize; i++) {
		error += (*c)[i] - (a[i] + b[i]);
		//error += (a[i] - b[i]);
	}
	gettimeofday(&t, NULL);
	printTime(&s, &t, "Check: ", "\n");
	fprintf(stdout, "Error: %.02f\n", error);
#endif

	gmacFree(a);
	gmacFree(b);

	return NULL;
}


int main(int argc, char *argv[])
{
	pthread_t *nThread;
	unsigned n = 0;

	setParam<unsigned>(&nIter, nIterStr, nIterDefault);
	setParam<size_t>(&vecSize, vecSizeStr, vecSizeDefault);

	vecSize = vecSize / nIter;

	nThread = (pthread_t *)malloc(nIter * sizeof(pthread_t));
	s = (float **)malloc(nIter * sizeof(float **));

	srand(time(NULL));

	for(n = 0; n < nIter; n++) {
		pthread_create(&nThread[n], NULL, addVector, &s[n]);
	}

	for(n = 0; n < nIter; n++) {
		pthread_join(nThread[n], NULL);
	}

	float error = 0;
	for(n = 0; n < nIter; n++) {
		for(int i = 0; i < vecSize; i++) {
			error += s[n][i] - 2;
		}
	}
	fprintf(stdout, "Total: %.02f\n", error);

	free(s);
	free(nThread);

}
