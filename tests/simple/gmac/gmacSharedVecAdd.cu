#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <gmac.h>

#include "utils.h"
#include "debug.h"

const char *nIterStr = "GMAC_NITER";
const char *vecSizeStr = "GMAC_VECSIZE";

const unsigned nIterDefault = 1;
const size_t vecSizeDefault = 1024 * 1024;

unsigned nIter = 0;
size_t vecSize = 0;
const size_t blockSize = 512;


static float *a, *b;
static struct param {
	int i;
	float *ptr;
} *param;


__global__ void vecAdd(float *c, float *a, float *b, size_t vecSize)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= vecSize) return;

	c[i] = a[i] + b[i];
}

void *addVector(void *ptr)
{
	gmactime_t s, t;
	struct param *p = (struct param *)ptr;
	gmacError_t ret = gmacSuccess;

	ret = gmacMalloc((void **)&p->ptr, vecSize * sizeof(float));
	assert(ret == gmacSuccess);

	// Call the kernel
	dim3 Db(blockSize);
	dim3 Dg(int(vecSize / blockSize));
	if(vecSize % blockSize) Dg.x++;

	getTime(&s);
	 vecAdd<<<Dg, Db>>>(gmacPtr((p->ptr)), gmacPtr(a + p->i * vecSize), gmacPtr(b + p->i * vecSize), vecSize);
	if(gmacThreadSynchronize() != gmacSuccess) CUFATAL();
	getTime(&t);
	printTime(&s, &t, "Run: ", "\n");

	getTime(&s);
	float error = 0;
	for(unsigned i = 0; i < vecSize; i++) {
		error += p->ptr[i] - (a[i + p->i * vecSize] + b[i + p->i * vecSize]);
		//error += (a[i] - b[i]);
	}
	getTime(&t);
	printTime(&s, &t, "Check: ", "\n");
	fprintf(stdout, "Error: %.02f\n", error);

    assert(error == 0);

	return NULL;
}


int main(int argc, char *argv[])
{
	thread_t *nThread;
	unsigned n = 0;
	gmacError_t ret = gmacSuccess;
	gmactime_t s, t;

	setParam<unsigned>(&nIter, nIterStr, nIterDefault);
	setParam<size_t>(&vecSize, vecSizeStr, vecSizeDefault);

	vecSize = vecSize / nIter;
	if(vecSize % nIter) vecSize++;

	nThread = (thread_t *)malloc(nIter * sizeof(thread_t));
	param = (struct param *)malloc(nIter * sizeof(struct param));

	getTime(&s);
	// Alloc & init input data
	ret = gmacGlobalMalloc((void **)&a, nIter * vecSize * sizeof(float));
	assert(ret == gmacSuccess);
	valueInit(a, 1.0, nIter * vecSize);
	ret = gmacGlobalMalloc((void **)&b, nIter * vecSize * sizeof(float));
	assert(ret == gmacSuccess);
	valueInit(b, 1.0, nIter * vecSize);

	// Alloc output data
	getTime(&t);
	printTime(&s, &t, "Alloc: ", "\n");

	for(n = 0; n < nIter; n++) {
		param[n].i = n;
		nThread[n] = thread_create(addVector, &(param[n]));
	}

	for(n = 0; n < nIter; n++) {
		thread_wait(nThread[n]);
	}

	gmacFree(a);
	gmacFree(b);

	float error = 0;
	for(n = 0; n < nIter; n++) {
		for(unsigned i = 0; i < vecSize; i++) {
			error += param[n].ptr[i] - 2;
		}
	}
	fprintf(stdout, "Total: %.02f\n", error);

	free(param);
	free(nThread);

    return error != 0;
}
