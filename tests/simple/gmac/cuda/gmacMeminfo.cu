#include <stdio.h>
#include <gmac/hip/hip_runtime.h>

const unsigned allocations  = 10;
const size_t allocationSize = 4 * 1024 * 1024;

int main(int argc, char *argv[])
{
    void *dummy[allocations];
    size_t freeMem;
    freeMem = gmacGetFreeMemory();
    fprintf(stdout, "Free memory: %zd\n", freeMem);

    for (unsigned i = 0; i < allocations; i++) {
        assert(gmacMalloc((void **)&dummy[i], allocationSize * sizeof(long)) == gmacSuccess);
        freeMem = gmacGetFreeMemory();
        fprintf(stdout, "Free memory: %zd\n", freeMem);
    }
    
    for (unsigned i = 0; i < allocations; i++) {
        assert(gmacFree(dummy[i]) == gmacSuccess);
        freeMem = gmacGetFreeMemory();
        fprintf(stdout, "Free memory: %zd\n", freeMem);
    }

    gmacMigrate(1);

    freeMem = gmacGetFreeMemory();
    fprintf(stdout, "Free memory: %zd\n", freeMem);

    for (unsigned i = 0; i < allocations; i++) {
        assert(gmacMalloc((void **)&dummy[i], allocationSize * sizeof(long)) == gmacSuccess);
        freeMem = gmacGetFreeMemory();
        fprintf(stdout, "Free memory: %zd\n", freeMem);
    }
    
    for (unsigned i = 0; i < allocations; i++) {
        assert(gmacFree(dummy[i]) == gmacSuccess);
        freeMem = gmacGetFreeMemory();
        fprintf(stdout, "Free memory: %zd\n", freeMem);
    }

    return 0;
}
