#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gmac/hip/hip_runtime.h>

__global__ void kernelFill(unsigned *A, unsigned off, size_t size)
{
    unsigned localIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx = localIdx + off;

    if (idx >= size) return;
    A[localIdx] = idx;
}

int main(int argc, char *argv[])
{
    const unsigned totalSize = 8 * 1024 * 1024;
    for (unsigned currentSize = totalSize; currentSize > 32; currentSize /= 2) {
        fprintf(stderr, "Testing object size %zd\n", currentSize);
        assert(totalSize % currentSize == 0);
        size_t nObjects = totalSize / currentSize;

        unsigned **objects = (unsigned **) malloc(nObjects * sizeof(int *));
        assert(objects != NULL);

        fprintf(stderr, "- Allocating: %zd objects\n", nObjects);
        for(size_t i = 0; i < nObjects; i++) {
            assert(gmacMalloc((void **)&objects[i], currentSize * sizeof(int)) == gmacSuccess);
        }

        fprintf(stderr, "- Running kernel\n");
        unsigned off = 0;

        dim3 Db(currentSize > 256? 256: currentSize);
        dim3 Dg(currentSize / Db.x);
        if (currentSize > 256 && currentSize % 256 != 0) Dg.x++;

        for(size_t i = 0; i < nObjects; i++) {
            kernelFill<<<Dg, Db>>>(gmacPtr(objects[i]), off, totalSize);
            off += currentSize;
        }
        gmacThreadSynchronize();

        fprintf(stderr, "- Checking\n");
        off = 0;
        for(size_t i = 0; i < nObjects; i++) {
            for(size_t j = 0; j < currentSize; j++) {
                size_t idx = off + j;
                assert(objects[i][j] == idx);
            }
            off += currentSize;
        }

        fprintf(stderr, "- Freeing: %zd objects\n", nObjects);
        for(size_t i = 0; i < nObjects; i++) {
            gmacFree(objects[i]);
        }

        free(objects);
    }

    return 0;
}
