#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gmac/hip/hip_runtime.h>

const size_t minCount = 1024;
const size_t maxCount = 2 * 1024 * 1024;

__global__ void null()
{
	return;
}

void init(long *ptr, int s, long v)
{
	for(int i = 0; i < s; i++) {
		ptr[i] = v;
	}
}

enum MemcpyType {
    GMAC_TO_GMAC = 1,
    HOST_TO_GMAC = 2,
    GMAC_TO_HOST = 3,
};

int memcpyTest(MemcpyType type, bool callKernel, void *(*memcpy_fn)(void *, const void *, size_t n))
{
    int error = 0;
    for (size_t count = minCount; count <= maxCount; count *= 2) {
        fprintf(stderr, "ALLOC: %zd\n", count * sizeof(long));
        long *baseSrc = (long *)malloc(count * sizeof(long));
        long *baseDst = (long *)malloc(count * sizeof(long));

        long *gmacSrc;
        long *gmacDst;

        if (type == GMAC_TO_GMAC) {
            assert(gmacMalloc((void **)&gmacSrc, count * sizeof(long)) == gmacSuccess);
            assert(gmacMalloc((void **)&gmacDst, count * sizeof(long)) == gmacSuccess);
        } else if (type == HOST_TO_GMAC) {
            gmacSrc = (long *)malloc(count * sizeof(long));
            assert(gmacMalloc((void **)&gmacDst, count * sizeof(long)) == gmacSuccess);
        } else if (type == GMAC_TO_HOST) {
            assert(gmacMalloc((void **)&gmacSrc, count * sizeof(long)) == gmacSuccess);
            gmacDst = (long *)malloc(count * sizeof(long));
        }

        for (size_t stride = 0, i = 1; stride < count/3; stride = i, i *= 2) {
            for (size_t copyCount = 0, j = 1; copyCount < count/3; copyCount = j, j *= 2) {
                init(baseSrc, int(count), 1);
                init(baseDst, int(count), 0);

                init(gmacSrc, int(count), 1);
                init(gmacDst, int(count), 0);
                assert(stride + copyCount <= count);

                if (callKernel) {
                    null<<<1, 1>>>();
                }
                assert(gmacThreadSynchronize() == gmacSuccess);
                memcpy   (baseDst + stride, baseSrc + stride, copyCount * sizeof(long));
                memcpy_fn(gmacDst + stride, gmacSrc + stride, copyCount * sizeof(long));

                int ret = memcmp(gmacDst, baseDst, count * sizeof(long));

                if (ret != 0) {
#if 0
                    fprintf(stderr, "Error: gmacToGmacTest size: %zd, stride: %zd, copy: %zd\n",
                            count     * sizeof(long),
                            stride    * sizeof(long),
                            copyCount * sizeof(long));
#endif
                    error = 1;
                    goto exit_test;
                }
#if 0
                for (unsigned k = 0; k < count; k++) {
                    int ret = baseDst[k] != gmacDst[k];
                    if (ret != 0) {
                        fprintf(stderr, "Error: gmacToGmacTest size: %zd, stride: %zd, copy: %zd. Pos %u\n", count     * sizeof(long),
                                stride    * sizeof(long),
                                copyCount * sizeof(long), k);
                        error = 1;
                    }
                }
#endif
            }
        }

        if (type == GMAC_TO_GMAC) {
            assert(gmacFree(gmacSrc) == gmacSuccess);
            assert(gmacFree(gmacDst) == gmacSuccess);
        } else if (type == HOST_TO_GMAC) {
            free(gmacSrc);
            assert(gmacFree(gmacDst) == gmacSuccess);
        } else if (type == GMAC_TO_HOST) {
            assert(gmacFree(gmacSrc) == gmacSuccess);
            free(gmacDst);
        }

        free(baseSrc);
        free(baseDst);
    }

exit_test:
    return error;
}

static void *gmacMemcpyWrapper(void *dst, const void *src, size_t size)
{
	return gmacMemcpy(dst, src, size);
}

int main(int argc, char *argv[])
{
    int           ret = memcpyTest(GMAC_TO_GMAC, false, gmacMemcpyWrapper);
    if (ret == 0) ret = memcpyTest(GMAC_TO_GMAC, true, gmacMemcpyWrapper);
    if (ret == 0) ret = memcpyTest(GMAC_TO_GMAC, false, memcpy);
    if (ret == 0) ret = memcpyTest(GMAC_TO_GMAC, true, memcpy);

    if (ret == 0) ret = memcpyTest(HOST_TO_GMAC, false, gmacMemcpyWrapper);
    if (ret == 0) ret = memcpyTest(HOST_TO_GMAC, true, gmacMemcpyWrapper);
    if (ret == 0) ret = memcpyTest(HOST_TO_GMAC, false, memcpy);
    if (ret == 0) ret = memcpyTest(HOST_TO_GMAC, true, memcpy);

    if (ret == 0) ret = memcpyTest(GMAC_TO_HOST, false, gmacMemcpyWrapper);
    if (ret == 0) ret = memcpyTest(GMAC_TO_HOST, true, gmacMemcpyWrapper);
    if (ret == 0) ret = memcpyTest(GMAC_TO_HOST, false, memcpy);
    if (ret == 0) ret = memcpyTest(GMAC_TO_HOST, true, memcpy);

    return ret;
}
