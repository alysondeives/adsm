#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gmac.h>

const size_t size = 4 * 1024 * 1024;
const size_t blockSize = 512;

__global__ void reset(long *a, long v)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= size) return;
	a[i] += v;
}

void init(long *ptr, int s, long v)
{
	for(int i = 0; i < s; i++) {
		ptr[i] = v;
	}
}

int check(long *ptr, int s)
{
	int a = 0;
	for(size_t i = 0; i < size; i++)
		a += ptr[i];
	return a - s;
}

int doTest(long *host, long *device, void *(*memcpy_fn)(void *, const void *, size_t n))
{
    init(host, size, 1);
    int ret1, ret2, ret3;

	// Call the kernel
	dim3 Db(blockSize);
	dim3 Dg(size / blockSize);
	if(size % blockSize) Db.x++;

	printf("Test full memcpy: ");
	memcpy_fn(device, host, size * sizeof(long));
	reset<<<Dg, Db>>>(gmacPtr(device), 1);
    gmacThreadSynchronize();
    ret1 = check(device, 2 * size);
	printf("%d\n", ret1);

	printf("Test partial memcpy: ");
	memcpy_fn(&device[size / 8], host, 3 * size / 4 * sizeof(long));
    ret2 = check(device, 5 * size / 4);
	printf("%d\n", ret2);

	fprintf(stderr,"Test reverse full: ");
	memcpy_fn(host, device, size * sizeof(long));
    ret3 = check(host, 5 * size / 4);
	fprintf(stderr, "%d\n", ret3);

    return (ret1 != 0 || ret2 != 0 || ret3 != 0);
}

static void *gmacMemcpyWrapper(void *dst, const void *src, size_t size)
{
	return gmacMemcpy(dst, src, size);
}

int main(int argc, char *argv[])
{
	long *ptr;
	long *host = (long *)malloc(size * sizeof(long));
	assert(host != NULL);

    // memcpy
	assert(gmacMalloc((void **)&ptr, size * sizeof(long)) == gmacSuccess);

    int res1 = doTest(host, ptr, memcpy);
    if (res1 != 0) fprintf(stderr, "Failed!\n");
	gmacFree(ptr);

    // gmacMemcpy
	assert(gmacMalloc((void **)&ptr, size * sizeof(long)) == gmacSuccess);
    int res2 = doTest(host, ptr, gmacMemcpyWrapper);
    if (res2 != 0) fprintf(stderr, "Failed!\n");
	gmacFree(ptr);

	free(host);

    return (res1 != 0 || res2 != 0);
}
