#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <gmac.h>
#include <gmac/hip/hip_runtime.h>

#include "debug.h"

const size_t vecSize = 1024 * 1024;
const size_t blockSize = 512;

const char *msg = "Done!";

__global__ void vecAdd(float *c, float *a, float *b)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= vecSize) return;

	c[i] = a[i] + b[i];
}


void randInit(float *a, size_t vecSize)
{
	for(int i = 0; i < vecSize; i++)
		a[i] = rand() / (float)RAND_MAX;
}


int main(int argc, char *argv[])
{
	float *a, *b, *c;

	srand(time(NULL));

	// Alloc & init input data
	if(gmacMalloc((void **)&a, vecSize * sizeof(float)) != gmacSuccess)
		CUFATAL();
	randInit(a, vecSize);
	if(gmacMalloc((void **)&b, vecSize * sizeof(float)) != gmacSuccess)
		CUFATAL();
	randInit(b, vecSize);

	// Alloc output data
	if(gmacMalloc((void **)&c, vecSize * sizeof(float)) != gmacSuccess)
		CUFATAL();

	// Call the kernel
	dim3 Db(blockSize);
	dim3 Dg(vecSize / blockSize);
	if(vecSize % blockSize) Db.x++;
	vecAdd<<<Dg, Db>>>(c, a, b);
	if(gmacThreadSynchronize() != gmacSuccess) CUFATAL();

	float error = 0;
	for(int i = 0; i < vecSize; i++) {
		error += c[i] - (a[i] + b[i]);
	}
	fprintf(stdout, "Error: %.02f\n", error);

	gmacFree(a);
	gmacFree(b);
	gmacFree(c);
}
