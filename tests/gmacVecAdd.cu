#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <gmac.h>

#include "utils.h"
#include "debug.h"


#define SIZE 1

const char *vecSizeStr = "GMAC_VECSIZE";
const size_t vecSizeDefault = 1024 * 1024;

size_t vecSize = 0;
const size_t blockSize = 512;

const char *msg = "Done!";

__global__ void vecAdd(float *c, float *a, float *b, size_t size)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= size) return;

	c[i] = a[i] + b[i];
}


int main(int argc, char *argv[])
{
	float *a = NULL, *b = NULL, *c = NULL;
	struct timeval s, t;

	setParam<size_t>(&vecSize, vecSizeStr, vecSizeDefault);
	fprintf(stdout, "Vector: %f\n", 1.0 * vecSize / 1024 / 1024);


	gettimeofday(&s, NULL);
	// Alloc & init input data
	if(gmacMalloc((void **)&a, vecSize * sizeof(float)) != gmacSuccess)
		CUFATAL();
	randInit(a, vecSize);
	if(gmacMalloc((void **)&b, vecSize * sizeof(float)) != gmacSuccess)
		CUFATAL();
	randInit(b, vecSize);
	// Alloc output data
	if(gmacMalloc((void **)&c, vecSize * sizeof(float)) != gmacSuccess)
		CUFATAL();
	gettimeofday(&t, NULL);
	printTime(&s, &t, "Alloc: ", "\n");

	// Call the kernel
	gettimeofday(&s, NULL);
	dim3 Db(blockSize);
	dim3 Dg(vecSize / blockSize);
	if(vecSize % blockSize) Dg.x++;
	vecAdd<<<Dg, Db>>>(gmacPtr(c), gmacPtr(a), gmacPtr(b), vecSize);
	if(gmacThreadSynchronize() != gmacSuccess) CUFATAL();
	gettimeofday(&t, NULL);
	printTime(&s, &t, "Run: ", "\n");

	gettimeofday(&s, NULL);
	float error = 0;
	for(int i = 0; i < vecSize; i++) {
		error += c[i] - (a[i] + b[i]);
	}
	gettimeofday(&t, NULL);
	printTime(&s, &t, "Check: ", "\n");

	fprintf(stderr,"Error: %f\n", error);

	gmacFree(a);
	gmacFree(b);
	gmacFree(c);

}
